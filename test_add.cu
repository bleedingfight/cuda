#include "inplaceOp.h"
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  using namespace std;
  int n = 32;
  int data_a[n] = {1};
  int data_b[n] = {2};
  int *d_a, *d_b;
  
  hipMalloc(&d_a, n * sizeof(int));
  hipMalloc(&d_b, n * sizeof(int));

  hipMemcpy(d_a, &data_a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &data_b, n * sizeof(int), hipMemcpyHostToDevice);

  inplaceMatrixAdd<<<1, n>>>(d_a, d_b, n);
  hipMemcpy(data_a, d_a,n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    cout << data_a[i] << endl;
}