#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include "inplaceOp.h"
#include <gtest/gtest.h>
using namespace std;

int main(int argc,char* argv[]) {
    testing::InitGoogleTest(&argc,argv);
    int dev = 0;
    hipSetDevice(dev);
    int nElems = 32;
    int size = nElems*sizeof(int);
    int *d_a,*d_b;
    int *result = new int[nElems];
    int* h_a = new int[nElems];
    int* h_b = new int[nElems];
    fill(h_a,h_a+nElems,1);
    fill(h_b,h_b+nElems,2);

    hipMalloc((float**) &d_a,size);
    hipMalloc((float**) &d_b,size);

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);

    dim3 block(nElems);
    dim3 grid (nElems/block.x);

    add_vector<<<grid,block>>>(d_a,d_b,nElems);
    
    
    hipMemcpy(result,d_a,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipDeviceReset();
    cout<<result[2]<<endl;
    EXPECT_EQ(result[2],3);
    hipFree(d_a);
    hipFree(d_b);
    // cudaDeviceDe
    

    delete [] h_a;
    delete [] h_b;
    delete [] result;
    return RUN_ALL_TESTS();
}
