#include "hip/hip_runtime.h"
#include "common.h"
void showDevice(const int devID) {
  hipDeviceProp_t deviceProp;
  hipError_t error_id = hipGetDeviceProperties(&deviceProp, devID);
  if (error_id != hipSuccess) {
    printf("CUDA Device Count Return %d Error:%s\n", (int)error_id,
           hipGetErrorString(error_id));
    exit(EXIT_FAILURE);
  }
  int driverVersion, runtimeVersion = 0;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("Current Device Name:%s\n", deviceProp.name);
  printf("CUDA Driver Version:%d.%d Runtime:%d.%d\n", driverVersion / 1000,
         (driverVersion % 100 / 10), runtimeVersion / 1000,
         (runtimeVersion % 100) / 10);
  printf("  Total amount of global memory:                 %.2f GBytes (%llu "
         "bytes)\n",
         (float)deviceProp.totalGlobalMem / pow(1024.0, 3),
         (unsigned long long)deviceProp.totalGlobalMem);
  printf("  GPU Clock rate:                                %.0f MHz (%0.2f "
         "GHz)\n",
         deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
  printf("  Memory Clock rate:                             %.0f Mhz\n",
         deviceProp.memoryClockRate * 1e-3f);
  printf("  Memory Bus Width:                              %d-bit\n",
         deviceProp.memoryBusWidth);

  if (deviceProp.l2CacheSize) {
    printf("  L2 Cache Size:                                 %d kb\n",
           deviceProp.l2CacheSize / 1024);
  }

  printf("  Max Texture Dimension Size (x,y,z)             1D=(%d), "
         "2D=(%d,%d), 3D=(%d,%d,%d)\n",
         deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
         deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
         deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
  printf("  Max Layered Texture Size (dim) x layers        1D=(%d) x %d, "
         "2D=(%d,%d) x %d\n",
         deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
         deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
         deviceProp.maxTexture2DLayered[2]);
  printf("  Total amount of constant memory:               %lu bytes\n",
         deviceProp.totalConstMem);
  printf("  Total amount of shared memory per block:       %lu bytes\n",
         deviceProp.sharedMemPerBlock);
  printf("  Total number of registers available per block: %d\n",
         deviceProp.regsPerBlock);
  printf("  Warp size:                                     %d\n",
         deviceProp.warpSize);
  printf("  Maximum number of threads per multiprocessor:  %d\n",
         deviceProp.maxThreadsPerMultiProcessor);
  printf("  Maximum number of threads per block:           %d\n",
         deviceProp.maxThreadsPerBlock);
  printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
         deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
         deviceProp.maxThreadsDim[2]);
  printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
         deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
         deviceProp.maxGridSize[2]);
  printf("  Maximum memory pitch:                          %lu bytes\n",
         deviceProp.memPitch);

  exit(EXIT_SUCCESS);
}
